#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optix_math.h>

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtBuffer<float4, 2>   output_buffer;
rtBuffer<float4, 2>   accumulation_buffer;
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(float,         normalization_const, , );

RT_PROGRAM void draw_result()
{
  output_buffer[launch_index] = accumulation_buffer[launch_index] / frame_number * normalization_const;
}

RT_PROGRAM void exception()
{
}
