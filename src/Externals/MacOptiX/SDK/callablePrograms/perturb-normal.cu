#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, x_frequency, , );
rtDeclareVariable(float, z_frequency, , );
rtDeclareVariable(float, amplitude, , );


RT_CALLABLE_PROGRAM float3 sine_adjust(float3 normal)
{
  float3 hit_point = ray.origin + t_hit * ray.direction;

  Onb onb(normal);

  float3 pn =
    onb.m_normal +
    onb.m_tangent * (amplitude * sinf(hit_point.x * x_frequency)) +
    onb.m_binormal * (amplitude * sinf(hit_point.z * z_frequency));

  return pn;
}

__global__ void sine_adjust_stub() {
  (void) sine_adjust( make_float3(0,0,0) );
}

