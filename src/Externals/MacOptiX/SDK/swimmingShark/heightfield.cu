#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;


rtDeclareVariable(float3,  boxmin, , );
rtDeclareVariable(float3,  boxmax, , );
rtDeclareVariable(float3,  cellsize, , );
rtDeclareVariable(float3,  inv_cellsize, , );
rtDeclareVariable(int2,    ncells, , );

rtBuffer<float, 2>  data;
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 

static __device__ __forceinline__ void computeNormal(int Lu, int Lv, float3 hitpos, float ya, float yb, float yc, float yd)
{
  float2 C = make_float2((hitpos.x - boxmin.x) * inv_cellsize.x,
                         (hitpos.z - boxmin.z) * inv_cellsize.z);
  float2 uv = C - make_float2(Lu, Lv);
  float dudx = inv_cellsize.x;
  float dvdz = inv_cellsize.z;
  float px = dudx*(yb + uv.y*yd);
  float pz = dvdz*(yc + uv.x*yd);
  shading_normal = geometric_normal = normalize(make_float3(-px, 1.0f, -pz));

  texcoord = (hitpos - boxmin) / (boxmax - boxmin);
  texcoord.y = texcoord.z;
  texcoord.z = 0.0f;
}

RT_PROGRAM void intersect(int primIdx)
{
  // Step 1 is setup (handled in CPU code)

  // Step 2 - transform ray into grid space and compute ray-box intersection
  float3 t0   = (boxmin - ray.origin)/ray.direction;
  float3 t1   = (boxmax - ray.origin)/ray.direction;
  float3 near = fminf(t0, t1);
  float3 far  = fmaxf(t0, t1);
  float tnear = fmaxf( near );
  float tfar  = fminf( far );

  if(tnear >= tfar)
    return;
  if(tfar < 1.e-6f)
    return;
  tnear = max(tnear, 0.f);
  tfar  = min(tfar,  ray.tmax);

  // Step 3
  size_t2 nnodes = data.size();
  float3 L = (ray.origin + tnear * ray.direction - boxmin) * inv_cellsize;
  int Lu = min(__float2int_rz(L.x), (unsigned int) (nnodes.x-2));
  int Lv = min(__float2int_rz(L.z), (unsigned int) (nnodes.y-2));

  // Step 4
  float3 D = ray.direction * inv_cellsize;
  int diu = D.x>0?1:-1;
  int div = D.z>0?1:-1;
  int stopu = D.x>0?(int)(nnodes.x)-1:-1;
  int stopv = D.z>0?(int)(nnodes.y)-1:-1;

  // Step 5
  float dtdu = abs(cellsize.x/ray.direction.x);
  float dtdv = abs(cellsize.z/ray.direction.z);

  // Step 6
  float far_u = (D.x>0.0f?Lu+1:Lu) * cellsize.x + boxmin.x;
  float far_v = (D.z>0.0f?Lv+1:Lv) * cellsize.z + boxmin.z;

  // Step 7
  float tnext_u = (far_u - ray.origin.x)/ray.direction.x;
  float tnext_v = (far_v - ray.origin.z)/ray.direction.z;

  // Step 8
  float yenter = ray.origin.y + tnear * ray.direction.y;
  while(tnear < tfar){
    float texit = min(tnext_u, tnext_v);
    float yexit = ray.origin.y + texit * ray.direction.y;

    // Step 9
    float d00 = data[make_uint2(Lu,   Lv)  ];
    float d01 = data[make_uint2(Lu,   Lv+1)];
    float d10 = data[make_uint2(Lu+1, Lv)  ];
    float d11 = data[make_uint2(Lu+1, Lv+1)];
    float datamin = min(min(d00, d01), min(d10, d11));
    float datamax = max(max(d00, d01), max(d10, d11));
    float ymin = min(yenter, yexit);
    float ymax = max(yenter, yexit);
    if(ymin <= datamax && ymax >= datamin){
      // Step 10
      float3 EC = (ray.origin + tnear * ray.direction - boxmin) * inv_cellsize - make_float3(Lu, 0.0f, Lv);
      EC.y = ray.origin.y + tnear * ray.direction.y;
      
      float ya = d00;
      float yb = d10-d00;
      float yc = d01-d00;
      float yd = d11-d10-d01+d00;
      float a = D.x*D.z*yd;
      float b = -D.y + D.x*yb + D.z*yc + (EC.x*D.z + EC.z*D.x)*yd;
      float c = ya - EC.y + EC.x*yb + EC.z*yc + EC.x*EC.z*yd;
      if(abs(a) < 1.e-6f){
        // Linear
        float tcell = -fdividef(c, b);
        float t = tnear + tcell;
        if(tcell > 0.0f && t < texit){
          if(rtPotentialIntersection( t )){
            computeNormal(Lu, Lv, ray.origin+t*ray.direction, ya, yb, yc, yd);
            if(rtReportIntersection(0))
              return;
          }
        }
      } else {
        // Solve quadatric
        b = -0.5f * b;
        float disc = b*b-a*c;
        if(disc > 0.0f){
          float root = sqrtf(disc);
          float tcell1 = fdividef(b + root, a);
          float t1 = tnear + tcell1;
          bool done = false;
          if(tcell1 >= 0.0f && t1 <= texit){
            if( rtPotentialIntersection( t1 ) ){
              computeNormal(Lu, Lv, ray.origin+t1*ray.direction, ya, yb, yc, yd);
              if(rtReportIntersection(0))
                done = true;
            }
          }
          float tcell2 = fdividef(b - root, a);
          float t2 = tnear + tcell2;
          if( tcell2 >= 0.0f && t2 <= texit){
            if( rtPotentialIntersection( t2 ) ) {
              computeNormal(Lu, Lv, ray.origin+t2*ray.direction, ya, yb, yc, yd);
              if(rtReportIntersection(0))
                done = true;
            }
          }
          if(done)
            return;
        }
      }
    }

    // Step 11
    yenter = yexit;
    if(tnext_u < tnext_v){
      Lu += diu;
      if(Lu == stopu)
        break;
      tnear = tnext_u;
      tnext_u += dtdu;
    } else {
      Lv += div;
      if(Lv == stopv)
        break;
      tnear = tnext_v;
      tnext_v += dtdv;
    }
  }
}


RT_PROGRAM void bounds (int, float result[6])
{
  optix::Aabb* aabb = (optix::Aabb*)result;
  aabb->set(boxmin, boxmax);
}
