#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "ppm.h"
#include "helpers.h"
#include "path_tracer.h"
#include "random.h"

using namespace optix;

//
// Ray generation program
//

rtDeclareVariable(rtObject,      top_object, , );
rtBuffer<float4, 2>              output_buffer;
rtBuffer<float4, 2>              debug_buffer;
rtBuffer<PackedPhotonRecord, 1>  photon_map;
rtBuffer<PackedHitRecord, 2>     rtpass_output_buffer;
rtBuffer<uint2, 2>               image_rnd_seeds;
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(float,         alpha, , );
rtDeclareVariable(float,         total_emitted, , );
rtDeclareVariable(float,         frame_number , , );
rtDeclareVariable(float3,        ambient_light , , );
rtDeclareVariable(uint,          use_debug_buffer, , );
rtDeclareVariable(PPMLight,      light , , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(ShadowPRD, shadow_prd, rtPayload, );


static __device__ __inline__ 
void accumulatePhoton( const PackedPhotonRecord& photon,
                       const float3& rec_normal,
                       const float3& rec_atten_Kd,
                       uint& num_new_photons, float3& flux_M )
{
  float3 photon_energy = make_float3( photon.c.y, photon.c.z, photon.c.w );
  float3 photon_normal = make_float3( photon.a.w, photon.b.x, photon.b.y );
  float p_dot_hit = dot(photon_normal, rec_normal);
  if (p_dot_hit > 0.01f) { // Fudge factor for imperfect cornell box geom
    float3 photon_ray_dir = make_float3( photon.b.z, photon.b.w, photon.c.x );
    float3 flux = photon_energy * rec_atten_Kd; // * -dot(photon_ray_dir, rec_normal);
    num_new_photons++;
    flux_M += flux;
  }
}

#if 0
#define check( condition, color ) \
{ \
  if( !(condition) ) { \
    debug_buffer[index] = make_float4( stack_current, node, photon_map_size, 0 ); \
    output_buffer[index] = make_color( color ); \
    return; \
  } \
}
#else
#define check( condition, color )
#endif


#define MAX_DEPTH 20 // one MILLION photons
RT_PROGRAM void gather()
{
  clock_t start = clock();
  PackedHitRecord rec = rtpass_output_buffer[launch_index];
  float3 rec_position = make_float3( rec.a.x, rec.a.y, rec.a.z );
  float3 rec_normal   = make_float3( rec.a.w, rec.b.x, rec.b.y );
  float3 rec_atten_Kd = make_float3( rec.b.z, rec.b.w, rec.c.x );
  uint   rec_flags    = __float_as_int( rec.c.y );
  float  rec_radius2  = rec.c.z;
  float  rec_photon_count = rec.c.w;
  float3 rec_flux     = make_float3( rec.d.x, rec.d.y, rec.d.z );
  float  rec_accum_atten = rec.d.w;

  // Check if this is hit point lies on an emitter or hit background 
  if( !(rec_flags & PPM_HIT) || rec_flags & PPM_OVERFLOW ) {
    output_buffer[launch_index] = make_float4(rec_atten_Kd);
    return;
  }

  unsigned int stack[MAX_DEPTH];
  unsigned int stack_current = 0;
  unsigned int node = 0; // 0 is the start

#define push_node(N) stack[stack_current++] = (N)
#define pop_node()   stack[--stack_current]

  push_node( 0 );

  int photon_map_size = photon_map.size(); // for debugging

  uint num_new_photons = 0u;
  float3 flux_M = make_float3( 0.0f, 0.0f, 0.0f );
  uint loop_iter = 0;
  do {

    check( node < photon_map_size, make_float3( 1,0,0 ) );
    PackedPhotonRecord& photon = photon_map[ node ];

    uint axis = __float_as_int( photon.d.x );
    if( !( axis & PPM_NULL ) ) {

      float3 photon_position = make_float3( photon.a );
      float3 diff = rec_position - photon_position;
      float distance2 = dot(diff, diff);

      if (distance2 <= rec_radius2) {
        accumulatePhoton(photon, rec_normal, rec_atten_Kd, num_new_photons, flux_M);
      }

      // Recurse
      if( !( axis & PPM_LEAF ) ) {
        float d;
        if      ( axis & PPM_X ) d = diff.x;
        else if ( axis & PPM_Y ) d = diff.y;
        else                      d = diff.z;

        // Calculate the next child selector. 0 is left, 1 is right.
        int selector = d < 0.0f ? 0 : 1;
        if( d*d < rec_radius2 ) {
          check( stack_current+1 < MAX_DEPTH, make_float3( 0,1,0) );
          push_node( (node<<1) + 2 - selector );
        }

        check( stack_current+1 < MAX_DEPTH, make_float3( 0,1,1) );
        node = (node<<1) + 1 + selector;
      } else {
        node = pop_node();
      }
    } else {
      node = pop_node();
    }
    loop_iter++;
  } while ( node );

  // Compute new N,R
  float R2 = rec_radius2;
  float N = rec_photon_count;
  float M = static_cast<float>( num_new_photons ) ;
  float new_N = N + alpha*M;
  rec.c.w = new_N;  // set rec.photon_count

  float reduction_factor2 = 1.0f;
  float new_R2 = R2;
  if( M != 0 ) {
    reduction_factor2 = ( N + alpha*M ) / ( N + M );
    new_R2 = R2*( reduction_factor2 ); 
    rec.c.z = new_R2; // set rec.radius2
  }

  // Compute indirectflux
  float3 new_flux = ( rec_flux + flux_M ) * reduction_factor2;
  rec.d = make_float4( new_flux ); // set rec.flux
  float3 indirect_flux = 1.0f / ( M_PIf * new_R2 ) * new_flux / total_emitted;

  // Compute direct
  float3 point_on_light;
  float dist_scale;
  if( light.is_area_light ) {
    uint2  seed   = image_rnd_seeds[launch_index];
    float2 sample = make_float2( rnd( seed.x ), rnd( seed.y ) ); 
    image_rnd_seeds[launch_index] = seed;
    point_on_light = light.anchor + sample.x*light.v1 + sample.y*light.v2; 
    dist_scale = 1.0f;
  } else {
    point_on_light = light.position;
    dist_scale = light.radius / ( M_PIf * 0.5f); 
  }
  float3 to_light    = point_on_light - rec_position;
  float  light_dist  = length( to_light );
  to_light = to_light / light_dist;
  float  n_dot_l     = fmaxf( 0.0f, dot( rec_normal, to_light ) );
  float  light_atten = n_dot_l;
  
  // TODO Should clip direct light to photon emiting code -- but we will ignore this for demo 
  //if( !light.is_area_light && acosf( dot( -to_light, light.direction )  ) > light.radius ) {
  //  light_atten = 0.0f;
  //}

  // PPM_IN_SHADOW will be set if this is a point light and we have already performed an occluded shadow query 
  if( rec_flags & PPM_IN_SHADOW ) {
    light_atten = 0.0f;
  }
  if ( light_atten > 0.0f ) {
    ShadowPRD prd;
    prd.attenuation = 1.0f;
    optix::Ray shadow_ray( rec_position, to_light, shadow_ray_type, scene_epsilon, light_dist - scene_epsilon );
    rtTrace( top_object, shadow_ray, prd );
    light_atten *= prd.attenuation * dot( -to_light, light.direction );
    rec.c.y = __int_as_float(  prd.attenuation == 0.0f && !light.is_area_light ? rec_flags|PPM_IN_SHADOW : rec_flags ); 
  } 
  light_atten /= dist_scale*light_dist*light_dist;
  if( light_atten < 0.0f ) light_atten = 0.0f;   // TODO Shouldnt be needed but we get acne near light w/out it
  rec.d.w = rec_accum_atten + light_atten;
  float avg_atten = rec.d.w / (frame_number+1.0f);
  float3 direct_flux = light.power * avg_atten *rec_atten_Kd;
  
  rtpass_output_buffer[launch_index] = rec;
  float3 final_color = direct_flux + indirect_flux + ambient_light*rec_atten_Kd; 
  output_buffer[launch_index] = make_float4(final_color);
  if(use_debug_buffer == 1)
    debug_buffer[launch_index] = make_float4( loop_iter, new_R2, new_N, M );
}

RT_PROGRAM void gather_any_hit()
{
  shadow_prd.attenuation = 0.0f;

  rtTerminateRay();
}


//
// Stack overflow program
//
rtDeclareVariable(float3, rtpass_bad_color, , );
RT_PROGRAM void gather_exception()
{
  output_buffer[launch_index] = make_float4(1.0f, 1.0f, 0.0f, 0.0f);
}


