#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include "distance_field.h"

using namespace optix;


// References:
// [1] Hart, J. C., Sandin, D. J., and Kauffman, L. H. 1989. Ray tracing deterministic 3D fractals
// [2] http://www.devmaster.net/forums/showthread.php?t=4448


rtDeclareVariable( float4, c4 , , );                // parameter quaternion
rtDeclareVariable( float,  alpha , , );
rtDeclareVariable( float,  delta , , );
rtDeclareVariable( float,  DEL , , );
rtDeclareVariable( float,  color_t , , );           // 0,1,2 are full colors, in between is morph
rtDeclareVariable( uint,   max_iterations , , );    // max iterations for divergence determination
rtDeclareVariable( float3, particle , , );          // position of force particle

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

// julia set object outputs this
rtDeclareVariable(float3, normal, attribute normal, ); 

// sphere outputs this
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 

struct PerRayData_radiance
{
  float3 result;
  float importance;
  int depth;
};

struct PerRayData_shadow
{
  float3 attenuation;
};

rtDeclareVariable(rtObject,                         top_object, , );
rtDeclareVariable(float, isect_t, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );


// Quaternion helpers.
static __host__ __device__ float4 mul( float4 a, float4 b )
{
  const float3 a2 = make_float3( a.y, a.z, a.w );
  const float3 b2 = make_float3( b.y, b.z, b.w );
  float3 r;
  r = a.x*b2 + b.x*a2 + cross( a2, b2 );
  return make_float4(
    a.x*b.x - dot( a2, b2 ),
    r.x, r.y, r.z );
}
static __host__ __device__ float4 square( float4 a )
{
  float3 a2 = make_float3( a.y, a.z, a.w );
  float3 r;
  r = 2*a.x*a2;
  return make_float4(
    a.x*a.x - dot( a2,a2 ),
    r.x, r.y, r.z );
}

// Intersect the bounding sphere of the Julia set.
static __host__ __device__ bool intersectBoundingSphere( float3 o, float3 d, float& tmin, float &tmax )
{
  const float sq_radius = 4.0f;
  const float b = dot( o, d );
  const float c = dot( o, o ) - sq_radius;
  const float disc = b*b - c;
  
  if( disc > 0.0f )
  {
    const float sdisc = sqrtf( disc );
    tmin = (-b - sdisc);
    tmax = (-b + sdisc);

    if(tmin > tmax)
    {
      const float temp = tmin;
      tmax = tmin;
      tmin = temp;
    }

    return true;
  } else {
    tmin = tmax = 0;
  }
  return false;
}

struct JuliaSet
{
  __host__ __device__
  JuliaSet(const unsigned int max_iterations):m_max_iterations(max_iterations)
  {}

  // Return the approximate lower bound on the distance from x to the set.
  __host__ __device__ __forceinline__
  float operator()( float3 x ) const
  {
    // Warp space around the particle to get the blob-effect.
    const float part_dist = length( particle - x );
    const float force = smoothstep( 0.0f, 1.0f, 0.1f / (part_dist*part_dist) ) * 0.2f;
    const float3 weg = (x-particle) / max(0.01f,part_dist);
    x -= weg * force;

    // Iterated values.
    float4 f_n  = make_float4( x, 0 );
    float4 fp_n = make_float4( 1, 0, 0, 0 );  // start derivative at real 1 (see [2]).

    const float sq_threshold = 16.0f;   // divergence threshold

    // Iterate to compute f_n and fp_n for the distance estimator.
    int i = m_max_iterations;
    while( i-- )
    {
      fp_n = 2.0f * mul( f_n, fp_n );   // z prime in [2]
      f_n = square( f_n ) + c4;         // equation (1) in [1]

      // Stop when we know the point diverges.
      // TODO: removing this condition burns 2 less registers and results in
      //       in a big perf improvement. Can we do something about it?
      if( dot(f_n,f_n) > sq_threshold )
        break;
    }

    // Distance estimation. Equation (8) from [1], with correction mentioned in [2].
    const float norm = length( f_n );
    const float julia_dist = 0.5f * norm * logf( norm ) / length( fp_n );

    return julia_dist;
    //return fminf( julia_dist, part_dist - 0.2f );  // this "renders" the particle as well
  }

  unsigned int m_max_iterations;
};


RT_PROGRAM void intersect(int primIdx)
{
  float tmin, tmax;
  if( intersectBoundingSphere(ray.origin, ray.direction, tmin, tmax) )
  {
    JuliaSet distance( max_iterations );

    // XXX inline the sphere tracing procedure here because nvcc isn't
    //     generating the right code i guess

    float3 ray_direction = ray.direction;
    float3 x = ray.origin + tmin * ray_direction;

    float dist_from_origin = tmin;

    // Compute epsilon using equation (16) of [1].
    //float epsilon = max(0.000001f, alpha * powf(dist_from_origin, delta));
    const float epsilon = 1e-3f;

    float dist = 0;
    for( unsigned int i = 0; i < 800; ++i )
    {
      dist = distance( x );

      // Step along the ray and accumulate the distance from the origin.
      x += dist * ray_direction;
      dist_from_origin += dist;

      // Check if we're close enough or too far.
      if( dist < epsilon || dist_from_origin > tmax  )
         break;
    }

    // Found intersection?
    if( dist < epsilon )
    {
      if( rtPotentialIntersection(dist_from_origin) )
      {
        // color HACK
        distance.m_max_iterations = 14;  // more iterations for normal estimate, to fake some more detail
        normal = estimate_normal(distance, x, DEL);
        rtReportIntersection( 0 );
      }
    }
  }
}

RT_PROGRAM void bounds (int, float result[6])
{
  optix::Aabb* aabb = (optix::Aabb*)result;
  const float sz = 1.4f;
  aabb->m_min = make_float3(-sz);
  aabb->m_max = make_float3(sz);
}


//
// Julia set shader.
//

RT_PROGRAM void julia_ch_radiance()
{
  const float3 p = ray.origin + isect_t * ray.direction;

  // ambient occlusion
  JuliaSet distance( max_iterations );
  float occlusion = 1.f;
  float fact = 7.f;
  const float delta = 0.05f;
  for( int i=0; i<4; ++i ) {
    const float dist = delta * i;
    occlusion -= fact * (dist - distance(p+dist*normal));
    fact *= 0.5f;
  }
  occlusion += 0.3f;
  occlusion *= occlusion;
  occlusion = clamp( occlusion, 0.2f, 1.0f );

  // base colors
  float3 red   = normal*0.5f + make_float3(0.5f);
  float3 green = red;
  float3 blue  = red;
  
  // red/orange
  red.x = abs(normal.x)*0.5f + 0.5f;
  red.x = max( red.x, 0.1f );
  red = red * make_float3( 0.8f, 0.1f+red.x, 0.1f );
  red.y += 0.2f * red.x;
  red.x += 0.6f;
  red.x *= max(occlusion,0.8f);
  red.y *= occlusion;
  red.z *= occlusion;
  
  // green
  green.x = abs(normal.x)*0.5f + 0.5f;
  green.z = -abs(normal.z)*0.5f + 0.5f;
  green.y = green.y * 0.7f + 0.3f;
  green = green * make_float3( 0.9f*green.y*green.y, 1.0f, 0.2f );
  green.x += 0.2f;
  green.x *= green.x;
  green.x *= occlusion;
  green.y = max(0.3f,green.y*occlusion);
  green.z *= occlusion;

  // blue
  blue.x = abs(normal.x)*0.5f + 0.5f;
  blue.y = abs(normal.y)*0.5f + 0.5f;
  blue.z = -abs(normal.z)*0.5f + 0.5f;
  blue.z = blue.z * 0.7f + 0.3f;
  blue.x += 0.2f;
  blue.y += 0.2f;
  blue = blue * make_float3( 0.9f*blue.y*blue.y, 1.0f*blue.z*blue.y, 1.0f );
  blue.z += 0.3f;
  blue.x *= blue.z * max(0.3f,occlusion);
  blue.y *= occlusion;
  blue.z = blue.z * max(0.6f,occlusion);

  // select color
  float3 c0 = green;
  float3 c1 = red;
  float ct = color_t;
  if( color_t > 1.0f ) {
    c0 = red;
    c1 = blue;
    ct -= 1.0f;
  }
  float3 result = dot(p,p) > ct*3.0f ? c0 : c1;

  // add glow close to particle
  const float part_dist = length( p-particle );
  const float glow = 1.0f - smoothstep( 0.0f, 1.0f, part_dist );
  result = result + make_float3(glow*0.4f);

  // add phong highlight
  const float3 l = make_float3( 1, 3, 1 );
  const float3 h = normalize( l - ray.direction );
  const float ndh = dot( normal, h );
  if( ndh > 0.0f ) {
    result = result + make_float3( 0.6f * occlusion * pow(ndh,20.0f) );
  }

  //result = make_float3( occlusion );

  // Reflection (disabled, doesn't look too great)
  /*
  PerRayData_radiance new_prd;             
  new_prd.importance = prd_radiance.importance;
  new_prd.depth = prd_radiance.depth + 1;
  if( prd_radiance.depth < 3 )
  {
    const float3 refl = reflect( ray.direction, normal );
    const Ray refl_ray = make_ray( p, refl, 0, 1e-3f, RT_DEFAULT_MAX );
    rtTrace( top_object, refl_ray, new_prd );
    result += new_prd.result * 0.1f;
  }
  */

  prd_radiance.result = result;
}

RT_PROGRAM void julia_ah_shadow()
{
  // this material is opaque, so it fully attenuates all shadow rays
  prd_shadow.attenuation = make_float3(0);
  rtTerminateRay();
}


//
// Chrome shader for force particle.
//

RT_PROGRAM void chrome_ch_radiance()
{
  float3 dir = ray.direction;
  const float3 hit_point = ray.origin + isect_t * dir;

  if( prd_radiance.depth < 3 )
  {
    PerRayData_radiance new_prd;             
    new_prd.importance = prd_radiance.importance;
    new_prd.depth = prd_radiance.depth + 1;
    
    const float3 refl = reflect( dir, shading_normal );
    const optix::Ray refl_ray = optix::make_Ray( hit_point, refl, 0, 1e-3f, RT_DEFAULT_MAX );
    rtTrace( top_object, refl_ray, new_prd );
    const float fresnel = fresnel_schlick( dot(shading_normal,-dir), 5.0f, 0.3f, 1.0f );
    const float diff = (shading_normal.y+1.f) * 0.5f;
    prd_radiance.result = new_prd.result * fresnel
      + make_float3(diff*diff*diff*0.1f);
  } else {
    prd_radiance.result = make_float3( 0 );
  }
}

RT_PROGRAM void chrome_ah_shadow()
{
  // this material is opaque, so it fully attenuates all shadow rays
  prd_shadow.attenuation = make_float3(0);
  rtTerminateRay();
}
