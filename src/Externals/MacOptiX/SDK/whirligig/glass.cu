#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"

using namespace optix;

rtDeclareVariable(rtObject,     top_object, , );
rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(int,          max_depth, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type, , );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

rtDeclareVariable(float,        importance_cutoff, , );
rtDeclareVariable(float3,       cutoff_color, , );
rtDeclareVariable(float,        fresnel_exponent, , );
rtDeclareVariable(float,        fresnel_minimum, , );
rtDeclareVariable(float,        fresnel_maximum, , );
rtDeclareVariable(float,        refraction_index, , );
rtDeclareVariable(int,          refraction_maxdepth, , );
rtDeclareVariable(int,          reflection_maxdepth, , );
rtDeclareVariable(float3,       refraction_color, , );
rtDeclareVariable(float3,       reflection_color, , );
rtDeclareVariable(float3,       extinction_constant, , );

struct PerRayData_radiance
{
  float3 result;
  float importance;
  int depth;
};

struct PerRayData_shadow
{
  float3 attenuation;
};

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

// -----------------------------------------------------------------------------

static __device__ __inline__ float3 TraceRay(float3 origin, float3 direction, int depth, float importance )
{
  optix::Ray ray = optix::make_Ray( origin, direction, radiance_ray_type, 0.0f, RT_DEFAULT_MAX );
  PerRayData_radiance prd;
  prd.depth = depth;
  prd.importance = importance;

  rtTrace( top_object, ray, prd );
  return prd.result;
}

static __device__ __inline__ float3 exp( const float3& x )
{
  return make_float3(exp(x.x), exp(x.y), exp(x.z));
}

// -----------------------------------------------------------------------------

RT_PROGRAM void closest_hit_radiance()
{
  // intersection vectors
  const float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); // normal
  const float3 fhp = rtTransformPoint(RT_OBJECT_TO_WORLD, front_hit_point);
  const float3 bhp = rtTransformPoint(RT_OBJECT_TO_WORLD, back_hit_point);
  const float3 i = ray.direction;                                            // incident direction
        float3 t;                                                            // transmission direction
        float3 r;                                                            // reflection direction

  float reflection = 1.0f;
  float3 result = make_float3(0.0f);
  
  const int depth = prd_radiance.depth;

  float3 beer_attenuation;
  if(dot(n, ray.direction) > 0) {
    // Beer's law attenuation
    beer_attenuation = exp(extinction_constant * t_hit);
  } else {
    beer_attenuation = make_float3(1);
  }

  // refraction
  if (depth < min(refraction_maxdepth, max_depth))
  {
    if ( refract(t, i, n, refraction_index) )
    {
      // check for external or internal reflection
      float cos_theta = dot(i, n);
      if (cos_theta < 0.0f)
        cos_theta = -cos_theta;
      else
        cos_theta = dot(t, n);

      reflection = fresnel_schlick(cos_theta, fresnel_exponent, fresnel_minimum, fresnel_maximum);

      float importance = prd_radiance.importance * (1.0f-reflection) * optix::luminance( refraction_color * beer_attenuation );
      float3 color = cutoff_color;
      if ( importance > importance_cutoff ) {
        color = TraceRay(bhp, t, depth+1, importance);
      }
      result += (1.0f - reflection) * refraction_color * color;
    }
    // else TIR
  } // else reflection==1 so refraction has 0 weight

  // reflection
  float3 color = cutoff_color;
  if (depth < min(reflection_maxdepth, max_depth))
  {
    r = reflect(i, n);
  
    float importance = prd_radiance.importance * reflection * optix::luminance( reflection_color * beer_attenuation );
    if ( importance > importance_cutoff ) {
      color = TraceRay( fhp, r, depth+1, importance );
    }
  }
  result += reflection * reflection_color * color;

  result = result * beer_attenuation;

  prd_radiance.result = result;
}

// -----------------------------------------------------------------------------

//
// Attenuates shadow rays for shadowing transparent objects
//
rtDeclareVariable(float3, shadow_attenuation, , );

RT_PROGRAM void any_hit_shadow()
{
  float3 world_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float nDi = fabs(dot(world_normal, ray.direction));

  prd_shadow.attenuation *= 1-fresnel_schlick(nDi, 5, 1-shadow_attenuation, make_float3(1));
  if(optix::luminance(prd_shadow.attenuation) < importance_cutoff)
    rtTerminateRay();
  else
    rtIgnoreIntersection();
}
