#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */


#include <optix.h>
#include <optix_math.h>
#include <optixu/optixu_matrix.h>
#include <optixu/optixu_aabb.h>
#include <helpers.h>
#include <sunsky.h>
#include <intersection_refinement.h>

/******************************************************************************\
 * 
 * Common variables and helper functions 
 *
\******************************************************************************/


struct PerRayData_radiance
{
  float3 result;
  float importance;
  int depth;
};


rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 


__device__ __inline__ float3 oceanQuerySkyModel( bool CEL, float3 ray_direction )
{
  const float d_dot_up = dot( ray_direction, sky_up );
  if( d_dot_up < 0.0f )
  {
    float3 clamped_dir = normalize( cross( ray_direction, sky_up ) );
    clamped_dir = normalize( cross( sky_up, clamped_dir ) );
    return querySkyModel( CEL, clamped_dir);
  }
  else
  {
    return querySkyModel( CEL, ray_direction );
  }
}   

/******************************************************************************\
 * 
 * Heightfield geometry programs
 *
\******************************************************************************/

rtDeclareVariable(float3,  boxmin, , );
rtDeclareVariable(float3,  boxmax, , );
rtDeclareVariable(float3,  cellsize, , );
rtDeclareVariable(float3,  inv_cellsize, , );
rtDeclareVariable(int2,    ncells, , );

rtBuffer<float,  2>  data;
rtBuffer<float4, 2>  normals;
rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );


__device__ float3 computeNormal( int Lu, int Lv, float3 hitpos )
{
  float2 C = make_float2((hitpos.x - boxmin.x) * inv_cellsize.x,
                         (hitpos.z - boxmin.z) * inv_cellsize.z);
  float2 uv = C - make_float2(Lu, Lv);

  float3 n00 = make_float3( normals[make_uint2(Lu,   Lv)  ] );
  float3 n01 = make_float3( normals[make_uint2(Lu,   Lv+1)] );
  float3 n10 = make_float3( normals[make_uint2(Lu+1, Lv)  ] );
  float3 n11 = make_float3( normals[make_uint2(Lu+1, Lv+1)] );

  return optix::bilerp( n00, n10, n01, n11, uv.x, uv.y ); 
}

RT_PROGRAM void intersect(int primIdx)
{
  // Step 1 is setup (handled in CPU code)

  // Step 2 - transform ray into grid space and compute ray-box intersection
  float3 t0   = (boxmin - ray.origin)/ray.direction;
  float3 t1   = (boxmax - ray.origin)/ray.direction;
  float3 near = fminf(t0, t1);
  float3 far  = fmaxf(t0, t1);
  float tnear = fmaxf( near );
  float tfar  = fminf( far );

  if(tnear >= tfar)
    return;
  if(tfar < 1.e-6f)
    return;
  tnear = max(tnear, 0.f);
  tfar  = min(tfar,  ray.tmax);

  // Step 3
  uint2 nnodes;
  nnodes.x = data.size().x;
  nnodes.y = data.size().y;
  float3 L = (ray.origin + tnear * ray.direction - boxmin) * inv_cellsize;
  int Lu = min(__float2int_rz(L.x), nnodes.x-2);
  int Lv = min(__float2int_rz(L.z), nnodes.y-2);

  // Step 4
  float3 D = ray.direction * inv_cellsize;
  int diu = D.x>0?1:-1;
  int div = D.z>0?1:-1;
  int stopu = D.x>0?(int)(nnodes.x)-1:-1;
  int stopv = D.z>0?(int)(nnodes.y)-1:-1;

  // Step 5
  float dtdu = abs(cellsize.x/ray.direction.x);
  float dtdv = abs(cellsize.z/ray.direction.z);

  // Step 6
  float far_u = (D.x>0.0f?Lu+1:Lu) * cellsize.x + boxmin.x;
  float far_v = (D.z>0.0f?Lv+1:Lv) * cellsize.z + boxmin.z;

  // Step 7
  float tnext_u = (far_u - ray.origin.x)/ray.direction.x;
  float tnext_v = (far_v - ray.origin.z)/ray.direction.z;

  // Step 8
  float yenter = ray.origin.y + tnear * ray.direction.y;
  while(tnear < tfar){
    float texit = min(tnext_u, tnext_v);
    float yexit = ray.origin.y + texit * ray.direction.y;

    // Step 9
    float d00 = data[make_uint2(Lu,   Lv)  ];
    float d01 = data[make_uint2(Lu,   Lv+1)];
    float d10 = data[make_uint2(Lu+1, Lv)  ];
    float d11 = data[make_uint2(Lu+1, Lv+1)];
    float datamin = min(min(d00, d01), min(d10, d11));
    float datamax = max(max(d00, d01), max(d10, d11));
    float ymin = min(yenter, yexit);
    float ymax = max(yenter, yexit);

    if(ymin <= datamax && ymax >= datamin) {

      float3 p00 = make_float3( boxmin.x + Lu*cellsize.x, d00, boxmin.z + Lv*cellsize.z );
      float3 p11 = make_float3( p00.x + cellsize.x,       d11, p00.z + cellsize.z ); 
      float3 p01 = make_float3( p00.x,                    d01, p11.z ); 
      float3 p10 = make_float3( p11.x,                    d10, p00.z ); 
      
      bool done = false;
      float3 n;
      float  t, beta, gamma;

      if( intersect_triangle( ray, p00, p11, p10, n, t, beta, gamma ) ) {
        if(rtPotentialIntersection(t)) {
          geometric_normal = normalize( n );
          shading_normal   = computeNormal( Lu, Lv, ray.origin+t*ray.direction );
          refine_and_offset_hitpoint( ray.origin + t*ray.direction, ray.direction,
                                      geometric_normal, p00,
                                      back_hit_point, front_hit_point );
          if(rtReportIntersection(0)) {
            done = true;
          }
        }
      }
      
      if( intersect_triangle( ray, p00, p01, p11, n, t, beta, gamma ) ) {
        if(rtPotentialIntersection(t)) {
          geometric_normal =  normalize( n );
          shading_normal   = computeNormal( Lu, Lv, ray.origin+t*ray.direction );
          refine_and_offset_hitpoint( ray.origin + t*ray.direction, ray.direction,
                                      geometric_normal, p00,
                                      back_hit_point, front_hit_point );

          if( rtReportIntersection( 0 ) ) {
            done = true;
          }
        }
      }
      if( done ) return;
    }

    // Step 11
    yenter = yexit;
    if(tnext_u < tnext_v){
      Lu += diu;
      if(Lu == stopu)
        break;
      tnear = tnext_u;
      tnext_u += dtdu;
    } else {
      Lv += div;
      if(Lv == stopv)
        break;
      tnear = tnext_v;
      tnext_v += dtdv;
    }
  }
}


RT_PROGRAM void bounds (int, float result[6])
{
  optix::Aabb* aabb = (optix::Aabb*)result;
  aabb->set(boxmin, boxmax);
}


/******************************************************************************\
 * 
 * Ocean water material programs 
 *
\******************************************************************************/

rtDeclareVariable(float3,       cutoff_color, , );
rtDeclareVariable(float,        fresnel_exponent, , );
rtDeclareVariable(float,        fresnel_minimum, , );
rtDeclareVariable(float,        fresnel_maximum, , );
rtDeclareVariable(float,        refraction_index, , );
rtDeclareVariable(float3,       refraction_color, , );
rtDeclareVariable(float3,       reflection_color, , );


RT_PROGRAM void closest_hit_radiance()
{
  const float3 i = ray.direction;     // incident direction

  float reflection = fresnel_maximum;
  float3 result = make_float3(0.0f);
  
  // refraction
  {
    float3 t = make_float3( 0.0f ); // transmission direction
    if ( refract(t, i, shading_normal, refraction_index) )
    {
      // check for external or internal reflection
      float cos_theta = dot(i, shading_normal);
      if (cos_theta < 0.0f) 
        cos_theta = -cos_theta;
      else 
        cos_theta = dot(t, shading_normal);

      reflection = fresnel_schlick(cos_theta, fresnel_exponent, fresnel_minimum, fresnel_maximum);
      if( dot( i, geometric_normal ) < 0.0f )
          result += (1.0f - reflection) * refraction_color * cutoff_color; 
      else
          result += (1.0f - reflection) * refraction_color * oceanQuerySkyModel( false, t );
    }
    // else TIR
  } 

  // reflection
  float3 color = cutoff_color;
  if( dot( i, geometric_normal ) < 0.0f )
  {
    float3 r = reflect(i, shading_normal);

    float importance = prd_radiance.importance *
                       reflection              *
                       optix::luminance( reflection_color );
    color = oceanQuerySkyModel( false, r );
  }

  result += reflection * reflection_color * color;

  prd_radiance.result = result;
}


/******************************************************************************\
 * 
 * Ocean sunsky miss program
 *
\******************************************************************************/



RT_PROGRAM void miss()
{
  prd_radiance.result = oceanQuerySkyModel( prd_radiance.depth == 0 , ray.direction );
}   
