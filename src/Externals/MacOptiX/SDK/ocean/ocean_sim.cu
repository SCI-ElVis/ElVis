#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 * * NVIDIA Corporation and its licensors retain all intellectual property and * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

///////////////////////////////////////////////////////////////////////////////

#include <optix.h>
#include <optix_math.h>
#include <hipfft/hipfft.h>
#include <hip/hip_math_constants.h>


rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(float, patch_size,, );
rtDeclareVariable(float, t,, );
rtBuffer<float2, 2>                    h0;
rtBuffer<float2, 2>                    ht;
rtBuffer<float2, 2>                    ik_ht;


/******************************************************************************\
 * 
 * Frequency space spectrum generation 
 * 
\******************************************************************************/
// complex math functions
__device__
float2 conjugate(float2 arg)
{ return make_float2(arg.x, -arg.y); }

__device__
float2 complex_exp(float arg)
{ return make_float2(cosf(arg), sinf(arg)); }

__device__
float2 complex_add(float2 a, float2 b)
{ return make_float2(a.x + b.x, a.y + b.y); }

__device__
float2 complex_mult(float2 ab, float2 cd)
{ return make_float2(ab.x * cd.x - ab.y * cd.y, ab.x * cd.y + ab.y * cd.x); }

RT_PROGRAM void generate_spectrum()
{
    unsigned int x = launch_index.x; 
    unsigned int y = launch_index.y;
    
    // calculate coordinates
    float2 k;
    k.x = HIP_PI_F * x / patch_size;
    k.y = 2.0f * HIP_PI_F * y / patch_size;

    // calculate dispersion w(k)
    float k_len = sqrtf(k.x*k.x + k.y*k.y);
    float w = sqrtf(9.81f * k_len);

    float2 h0_k  = h0[ make_uint2( x, y ) ];
    float2 h0_mk = h0[make_uint2( x, launch_dim.y-1-y )];

    float2 h_tilda = complex_add( complex_mult(h0_k, complex_exp(w * t)),
                                  complex_mult(conjugate(h0_mk), complex_exp(-w * t)) );
    float2 ik_h_tilda = k*h_tilda;

    ht[ launch_index ] = h_tilda;
    ik_ht[ launch_index ] = ik_h_tilda;
}


/******************************************************************************\
 * 
 * Normal calculation 
 * 
\******************************************************************************/
rtBuffer<float,  2>                    data;
rtBuffer<float4, 2>                    normals;

rtDeclareVariable(float, height_scale, , );

RT_PROGRAM void calculate_normals()
{
    unsigned int x = launch_index.x; 
    unsigned int y = launch_index.y;
    unsigned int width  = launch_dim.x;
    unsigned int height = launch_dim.y;

    float2 slope;
    if ( (x > 0u) && ( y > 0u ) && ( x < width-1u ) && ( y < height-1u ) ) {
      slope.x = data[ make_uint2( x+1, y   ) ]- data[ make_uint2( x-1, y   ) ];
      slope.y = data[ make_uint2( x,   y+1 ) ]- data[ make_uint2( x,   y-1 ) ];
    } else {
      slope = make_float2(0.0f, 0.0f);
    }
    float3 normal = normalize( cross( make_float3( 0.0f,          slope.y*height_scale, 2.0f / width ),
                                      make_float3( 2.0f / height, slope.x*height_scale, 0.0f         ) ) );
    normals[launch_index] = make_float4( normal, 0.0f );
}

